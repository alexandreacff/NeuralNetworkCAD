
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ double sigmoid(double x){
	return 1.0f / (1.0f + exp(-x));
}

__device__ double dSigmoid(double x){
	return x * (1.0f - x); 
}

__global__ void forwardFeed(double* inputLayer, double* hiddenWeights, double* hiddenLayer, double* outputLayer, double* outputWeights, double* outputLayerBias, double* hiddenLayerBias, int numHiddenNodes, int numInputs, int numOutputs, int trainingSetIndex) {
    int i = trainingSetIndex;

    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < numHiddenNodes; j += blockDim.x * gridDim.x) {
        double activation = hiddenLayerBias[j];
        for (int k = 0; k < numInputs; k++) {
            activation += inputLayer[(i * numInputs) + k] * hiddenWeights[(j * numInputs) + k];
        }
        hiddenLayer[j] = sigmoid(activation);
    }

    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < numOutputs; j += blockDim.x * gridDim.x) {
        double activation = outputLayerBias[j];
        for (int k = 0; k < numHiddenNodes; k++) {
            activation += hiddenLayer[k] * outputWeights[j + (k * numOutputs)];
        }
        outputLayer[j] = sigmoid(activation);
    }
}

__global__ void backpropagate(double* trainingInputs, double* hiddenLayer, double* hiddenWeights, double* outputLayer, double* outputWeights, double* trainingOutputs, double* hiddenLayerBias, double* outputLayerBias, int numHiddenNodes, int numInputs, int numOutputs, int numTrainingSets, double lr) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numTrainingSets) {
        int i = tid;

        double deltaOutput[1];

	//Calcula o Mean Squared Error (MSE) dos pesos do output
        for(int j = 0; j < numOutputs; j++){
           double dError = (trainingOutputs[i * numOutputs + j] - outputLayer[j]);
           deltaOutput[j] = dError * dSigmoid(outputLayer[j]);
        }

        double deltaHidden[4];
	//Calcula o MSE para o erro das camadas ocultas 
        for(int j = 0; j < numHiddenNodes; j++){
           double dError = 0.0f; 
           for(int k = 0; k < numOutputs; k++){
               dError += deltaOutput[k] * outputWeights[(j * 1) + k]; 
           }
           deltaHidden[j] = dError * dSigmoid(hiddenLayer[j]); 
        }

	//Aplica as mudanças dos pesos do output 
        for(int j = 0; j < numOutputs; j++){
           outputLayerBias[j] += deltaOutput[j] * lr;
           for(int k = 0; k < numHiddenNodes; k++){
               outputWeights[(k * numOutputs) + j] += hiddenLayer[k] * deltaOutput[j] * lr;
           }
        }

       //Aplicar as mudanças em pesos ocultos
       for(int j = 0; j < numHiddenNodes; j++){
           hiddenLayerBias[j] += deltaHidden[j] * lr; 
           for(int k = 0; k < numInputs; k++){
               hiddenWeights[(k * numOutputs) + j] += trainingInputs[(i * numInputs) + k] * deltaHidden[j] * lr; 
           }
       }
   }
}
